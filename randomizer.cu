#include "hip/hip_runtime.h"
#include "randomizer.h"
#include "flow.h"
#include "worker.h"
#include "generator.h"




double randomizerGpu (unsigned *vec, unsigned length, unsigned min, unsigned max) {


  double middleValue = 0.0;


  unsigned *minGpu;
  unsigned *maxGpu;
  unsigned *vecGpu;
  unsigned *lengthGpu;
  unsigned countFlow = getCountFlow();
  unsigned *countFlowGpu;
  unsigned countBlock = getCountBlock();


  unsigned *poolA = createPoolA(countFlow);
  unsigned *poolB = createPoolB(poolA, countFlow);
  unsigned *poolX = createPoolX(poolA, countFlow);


  unsigned *poolAGpu;
  unsigned *poolBGpu;
  unsigned *poolXGpu;


  double *poolMiddleValue = new double[countFlow];
  double *poolMiddleValueGpu;


  unsigned lastA = getLastA(poolA, countFlow);
  unsigned *lastAGpu;


  hipMalloc((void **)&vecGpu, length * sizeof(unsigned));
  hipMalloc((void **) &poolAGpu, countFlow * sizeof(unsigned));
  hipMalloc((void **) &poolBGpu, countFlow * sizeof(unsigned));
  hipMalloc((void **) &poolXGpu, countFlow * sizeof(unsigned));
  hipMalloc((void **) &poolMiddleValueGpu, countFlow * sizeof(double));
  hipMalloc((void **) &countFlowGpu, sizeof(unsigned));
  hipMalloc((void **) &lengthGpu, sizeof(unsigned));
  hipMalloc((void **) &minGpu, sizeof(unsigned));
  hipMalloc((void **) &maxGpu, sizeof(unsigned));
  hipMalloc((void **) &lastAGpu, sizeof(unsigned));


  //hipMemcpy(vecGpu, vec, length * sizeof(unsigned), hipMemcpyHostToDevice);
  hipMemcpy(poolAGpu, poolA, countFlow * sizeof(unsigned), hipMemcpyHostToDevice);
  hipMemcpy(poolBGpu, poolB, countFlow * sizeof(unsigned), hipMemcpyHostToDevice);
  hipMemcpy(poolXGpu, poolX, countFlow * sizeof(unsigned), hipMemcpyHostToDevice);
  hipMemcpy(poolMiddleValueGpu, poolMiddleValue, countFlow * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(lengthGpu, &length, sizeof(unsigned), hipMemcpyHostToDevice);
  hipMemcpy(countFlowGpu, &countFlow, sizeof(unsigned), hipMemcpyHostToDevice);
  hipMemcpy(minGpu, &min, sizeof(unsigned), hipMemcpyHostToDevice);
  hipMemcpy(maxGpu, &max, sizeof(unsigned), hipMemcpyHostToDevice);
  hipMemcpy(lastAGpu, &lastA, sizeof(unsigned), hipMemcpyHostToDevice);


  worker<<<countBlock, countFlow>>>(vecGpu, poolBGpu, poolXGpu, countFlowGpu, lengthGpu, poolMiddleValueGpu, minGpu, maxGpu, lastAGpu);


  hipMemcpy(poolMiddleValue, poolMiddleValueGpu, countFlow * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(vec, vecGpu, length * sizeof(unsigned), hipMemcpyDeviceToHost);


  for(unsigned i = 0; i < countFlow; i++) {
    middleValue += poolMiddleValue[i];
  }


  middleValue = middleValue / length;

  delete [] poolA;
  delete [] poolX;
  delete [] poolMiddleValue;
  hipFree(vecGpu);
  hipFree(poolAGpu);
  hipFree(poolXGpu);
  hipFree(poolMiddleValueGpu);
  hipFree(lengthGpu);
  hipFree(minGpu);
  hipFree(maxGpu);
  hipFree(countFlowGpu);
  


  return middleValue;;
}
