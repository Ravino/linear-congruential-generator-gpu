#include <iostream>
    #include <hip/hip_runtime_api.h>


int main()
    {
  int deviceCount;
  hipDeviceProp_t deviceProp;

  //Сколько устройств CUDA установлено на PC.
  hipGetDeviceCount(&deviceCount);
  std::cout << "Device count: " <<  deviceCount << "\n\n";


  for (int i = 0; i < deviceCount; i++)
      {

    //Получаем информацию об устройстве
    hipGetDeviceProperties(&deviceProp, i);


    //Выводим иформацию об устройстве
    std::cout << "Device name: " << deviceProp.name << "\n";
    std::cout << "Total global memory: " << deviceProp.totalGlobalMem << "\n";
    std::cout << "Shared memory per block: " << deviceProp.sharedMemPerBlock << "\n";
    std::cout << "Registers per block: " << deviceProp.regsPerBlock << "\n";
    std::cout << "Warp size: " << deviceProp.warpSize << "\n";
    std::cout << "Memory pitch: " << deviceProp.memPitch << "\n";
    std::cout << "Max threads per block: " << deviceProp.maxThreadsPerBlock << "\n";
    std::cout << "Max threads dimensions: x = " << deviceProp.maxThreadsDim[0] << ", y = " << deviceProp.maxThreadsDim[1] << ", z = " << deviceProp.maxThreadsDim[2] << "\n";
    std::cout << "Max grid size: x = " << deviceProp.maxGridSize[0] << ", y = " << deviceProp.maxGridSize[1] << ", z = " << deviceProp.maxGridSize[2] << "\n";
    std::cout << "Clock rate: " << deviceProp.clockRate << "\n";
    std::cout << "Total constant memory: " << deviceProp.totalConstMem << "\n";
    std::cout << "Compute capability: " << deviceProp.major << " " << deviceProp.minor << "\n";
    std::cout << "Texture alignment: " << deviceProp.textureAlignment << "\n";
    std::cout << "Device overlap: " << deviceProp.deviceOverlap << "\n";
    std::cout << "Multiprocessor count: " << deviceProp.multiProcessorCount << "\n";
    std::cout << "Kernel execution timeout enabled: " << deviceProp.kernelExecTimeoutEnabled ? "true \n" : "false \n";
  }

  return 0;
}
