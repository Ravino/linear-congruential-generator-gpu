#include "hip/hip_runtime.h"
#include "worker.h"


__global__ void worker (unsigned *vecGpu, unsigned *poolBGpu, unsigned *poolXGpu, unsigned *countFlowGpu, unsigned *lengthGpu, double *poolMiddleValueGpu, unsigned *minGpu, unsigned *maxGpu, unsigned *lastAGpu) {

  double currentMiddleValue = 0.0;


  unsigned flow = threadIdx.x;


  unsigned b = poolBGpu[flow];
  unsigned x = poolXGpu[flow];


  unsigned lastA = *lastAGpu;
  unsigned countFlow = *countFlowGpu;
  unsigned length = *lengthGpu;
  unsigned min = *minGpu;
  unsigned max = *maxGpu;


  for (unsigned i = flow; i < length; i + countFlow) {
    vecGpu[i] = (x % (max - min)) + min;
    currentMiddleValue += (double)vecGpu[i];
    x = x * lastA + b;;
  }


  poolMiddleValueGpu[flow] = currentMiddleValue;
}
